#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <chrono>
#include "ProjectDefinitions.h"
using namespace std::chrono;

#ifdef USE_CUDA

#define MAXIMUM_NUMBER_OF_THREADS						1024
#define MAXIMUM_NUMBER_OF_THREADS_COMPARING				500
#define MAXIMUM_NUMBER_OF_THREADS_DRIFT_HANDLING		1024

__global__ void runChannelFilterGPU(
	float* d_result,
	float* d_intermediateResult,
	float* d_signal,
	float* d_coeffsA,
	float* d_coeffsB,
	uint16_t signalWidth,
	uint32_t signalLength)
{
	// Forward filtering
	//int x = blockIdx.x; // counts the channels (width)
	uint16_t x = threadIdx.x; // counts the channels (width)

	for (int i = 0; i < signalLength; i++)
	{
		uint32_t index = ((i*signalWidth) + x);
		float tmp = 0.f;
		d_intermediateResult[index] = 0.f;
		for (int16_t j = 0; j < (int16_t)NUMBER_OF_B_COEFF; j++)
		{
			// Every second b coefficient is 0.
			if ((i - (j * 2)) < 0) continue;
			tmp += d_coeffsB[j] * d_signal[index - (j * 2)*signalWidth];
		}


		for (int16_t j = 0; j < (int16_t)NUMBER_OF_A_COEFF; j++)
		{
			// The first a coefficient is 1.
			if ((i - (j + 1)) < 0) continue;
			tmp -= d_coeffsA[j] * d_intermediateResult[index - (j + 1)*signalWidth];
		}

		d_intermediateResult[index] = tmp;
	}

	//x = (gridDim.x - 1) - blockIdx.x;
	//x = (blockDim.x - 1) - threadIdx.x;

	// Reverse filtering

	for (int i = signalLength - 1; i >= 0; i--)
	{
		uint32_t index = ((i*signalWidth) + x);
		float tmp = 0.;
		d_result[index] = 0.f;
		for (int16_t j = 0; j < (int16_t)NUMBER_OF_B_COEFF; j++)
		{
			// Every second b coefficient is 0.
			if ((i + (j * 2)) > (signalLength - 1)) continue;
			tmp += d_coeffsB[j] * d_intermediateResult[(index)+(j * 2)*signalWidth];
		}

		for (int16_t j = 0; j < (int16_t)NUMBER_OF_A_COEFF; j++)
		{
			// The first a coefficient is 1.
			if ((i + (j + 1)) > (signalLength - 1)) continue;
			tmp -= d_coeffsA[j] * d_result[(index)+(j + 1)*signalWidth];
		}

		d_result[index] = tmp;
	}
}

__global__ void runChannelFilterForwardGPU(
	float* d_intermediateResult,
	float* d_signal,
	float* d_coeffsA,
	float* d_coeffsB,
	uint16_t signalWidth,
	uint32_t signalLength)
{
	// Forward filtering
	//int x = blockIdx.x; // counts the channels (width)
	uint16_t x = threadIdx.x; // counts the channels (width)

	/*
	for (int i = 0; i < signalLength; i++)
	{
		uint32_t index = ((i*signalWidth) + x);
		d_intermediateResult[index] = 0.f;
	}
	__syncthreads();
	*/

	for (int i = 0; i < signalLength; i++)
	{
		uint32_t index = ((i*signalWidth) + x);
		float tmp = 0.f;
		d_intermediateResult[index] = 0.f;
		for (int16_t j = 0; j < (int16_t)NUMBER_OF_B_COEFF; j++)
		{
			// Every second b coefficient is 0.
			if ((i - (j * 2)) < 0) continue;
			tmp += d_coeffsB[j] * d_signal[index - (j * 2)*signalWidth];
		}


		for (int16_t j = 0; j < (int16_t)NUMBER_OF_A_COEFF; j++)
		{
			// The first a coefficient is 1.
			if ((i - (j + 1)) < 0) continue;
			tmp -= d_coeffsA[j] * d_intermediateResult[index - (j + 1)*signalWidth];
		}

		d_intermediateResult[index] = tmp;
	}
}

__global__ void runChannelFilterReverseGPU(
	float* d_result,
	float* d_intermediateResult,
	float* d_coeffsA,
	float* d_coeffsB,
	uint16_t signalWidth,
	uint32_t signalLength)
{
	// Forward filtering
	//int x = blockIdx.x; // counts the channels (width)
	uint16_t x = threadIdx.x; // counts the channels (width)

	//x = (gridDim.x - 1) - blockIdx.x;
	//x = (blockDim.x - 1) - threadIdx.x;
	// Reverse filtering
	/*
	for (int i = 0; i < signalLength; i++)
	{
		uint32_t index = ((i*signalWidth) + x);
		d_result[index] = 0.f;
	}
	__syncthreads();
	*/

	for (int i = signalLength - 1; i >= 0; i--)
	{
		uint32_t index = ((i*signalWidth) + x);
		float tmp = 0.;
		d_result[index] = 0.f;
		for (int16_t j = 0; j < (int16_t)NUMBER_OF_B_COEFF; j++)
		{
			// Every second b coefficient is 0.
			//if ((i + (j * 2)) > (signalLength - 1)) continue;
			if ((index + (j * 2)*signalWidth) >= (signalLength*signalWidth)) continue;
			tmp += d_coeffsB[j] * d_intermediateResult[index + (j * 2)*signalWidth];
		}

		for (int16_t j = 0; j < (int16_t)NUMBER_OF_A_COEFF; j++)
		{
			// The first a coefficient is 1.
			//if ((i + (j + 1)) > (signalLength - 1)) continue;
			if ((index + (j + 1)*signalWidth) >= (signalLength*signalWidth)) continue;
			tmp -= d_coeffsA[j] * d_result[index + (j + 1)*signalWidth];
		}

		d_result[index] = tmp;
	}
}

__global__ void runFilterReplicateGPU(
	float* d_result,
	const float* d_Signal,
	const float* d_filterKernel,
	uint16_t kernelDim,
	uint32_t signalLength,
	uint16_t signalWidth)
{
	// Perform filtering

	// setup variables
	uint16_t kernelHalfSize = kernelDim / 2;
	uint32_t y = threadIdx.y; // counts the channels (width)
	uint32_t x = threadIdx.x + blockDim.x*blockIdx.x; // counts the number of samples
	float tmpFilterValue = 0;

	if (x < signalLength)
	{
		// If away from border
		if (x >= kernelHalfSize && y >= kernelHalfSize && ((signalLength - 1) - x) >= kernelHalfSize && ((signalWidth - 1) - y) >= kernelHalfSize)
		{
			// for each location apply the filter kernel
			for (uint32_t i = 0; i < kernelDim; i++) // assumes kernel af uneven squared size
			{
				for (uint32_t j = 0; j < kernelDim; j++)
				{
					tmpFilterValue += d_Signal[((((x - 1) + i)*signalWidth) + (y - 1)) + j] * d_filterKernel[j + (i*kernelDim)];
				}
			}
		}
		else // Close to border
		{
			uint32_t imageStarti = 0;
			uint32_t imageStartj = 0;
			uint32_t imageStartx = x;
			uint32_t imageStarty = y;
			uint32_t kernelIMax = kernelDim;
			uint32_t kernelJMax = kernelDim;
			uint32_t extraSubtractI = 0;
			uint32_t extraSubtractY = 0;

			// find startlocations
			bool corner = false;

			if (x < kernelHalfSize && y < kernelHalfSize) // corner ⌈
			{
				tmpFilterValue += d_Signal[(x*signalWidth) + y] * d_filterKernel[0];
				tmpFilterValue += d_Signal[(x*signalWidth) + y] * d_filterKernel[1];
				tmpFilterValue += d_Signal[(x*signalWidth) + y + 1] * d_filterKernel[2];
				tmpFilterValue += d_Signal[(x*signalWidth) + y] * d_filterKernel[3];
				tmpFilterValue += d_Signal[((x + 1)*signalWidth) + y] * d_filterKernel[6];
				corner = true;
			}

			if (y < kernelHalfSize && ((signalLength - 1) - x) < kernelHalfSize) // corner ⌉
			{
				tmpFilterValue += d_Signal[(x*signalWidth) + y] * d_filterKernel[1];
				tmpFilterValue += d_Signal[(x*signalWidth) + y] * d_filterKernel[2];
				tmpFilterValue += d_Signal[(x*signalWidth) + y] * d_filterKernel[5];
				tmpFilterValue += d_Signal[(x*signalWidth) + (y + 1)] * d_filterKernel[8];
				tmpFilterValue += d_Signal[((x - 1)*signalWidth) + y] * d_filterKernel[0];
				corner = true;
			}

			if (x < kernelHalfSize && ((signalWidth - 1) - y) < kernelHalfSize) // corner ⌊
			{
				tmpFilterValue += d_Signal[(x*signalWidth) + y] * d_filterKernel[3];
				tmpFilterValue += d_Signal[(x*signalWidth) + y] * d_filterKernel[6];
				tmpFilterValue += d_Signal[(x*signalWidth) + y] * d_filterKernel[7];
				tmpFilterValue += d_Signal[(x*signalWidth) + (y - 1)] * d_filterKernel[0];
				tmpFilterValue += d_Signal[((x + 1)*signalWidth) + y] * d_filterKernel[8];
				corner = true;
			}

			if (((signalLength - 1) - x) < kernelHalfSize && ((signalWidth - 1) - y) < kernelHalfSize) // corner ⌋
			{
				tmpFilterValue += d_Signal[(x*signalWidth) + y] * d_filterKernel[5];
				tmpFilterValue += d_Signal[(x*signalWidth) + y] * d_filterKernel[7];
				tmpFilterValue += d_Signal[(x*signalWidth) + y] * d_filterKernel[8];
				tmpFilterValue += d_Signal[(x*signalWidth) + (y - 1)] * d_filterKernel[2];
				tmpFilterValue += d_Signal[((x - 1)*signalWidth) + y] * d_filterKernel[6];
				corner = true;
			}

			if (x < kernelHalfSize)
			{
				extraSubtractI = kernelHalfSize;
				imageStarti = kernelHalfSize;
				imageStartx = kernelHalfSize;

				if (!corner)
				{
					tmpFilterValue += d_Signal[(x*signalWidth) + (y - 1)] * d_filterKernel[0];
					tmpFilterValue += d_Signal[(x*signalWidth) + y] * d_filterKernel[3];
					tmpFilterValue += d_Signal[(x*signalWidth) + (y + 1)] * d_filterKernel[6];
				}

			}

			if (y < kernelHalfSize)
			{
				extraSubtractY = kernelHalfSize;
				imageStartj = kernelHalfSize;
				imageStarty = kernelHalfSize;

				if (!corner)
				{
					tmpFilterValue += d_Signal[((x - 1)* signalWidth) + y] * d_filterKernel[0];
					tmpFilterValue += d_Signal[(x*signalWidth) + y] * d_filterKernel[1];
					tmpFilterValue += d_Signal[((x + 1)* signalWidth) + y] * d_filterKernel[2];
				}
			}

			if (((signalLength - 1) - x) < kernelHalfSize)
			{
				kernelIMax = kernelDim - kernelHalfSize;

				if (!corner)
				{
					tmpFilterValue += d_Signal[(x* signalWidth) + (y - 1)] * d_filterKernel[2];
					tmpFilterValue += d_Signal[(x* signalWidth) + y] * d_filterKernel[5];
					tmpFilterValue += d_Signal[(x* signalWidth) + (y + 1)] * d_filterKernel[8];
				}
			}

			if (((signalWidth - 1) - y) < kernelHalfSize)
			{
				kernelJMax = kernelDim - kernelHalfSize;

				if (!corner)
				{
					tmpFilterValue += d_Signal[((x - 1)* signalWidth) + y] * d_filterKernel[6];
					tmpFilterValue += d_Signal[(x*signalWidth) + y] * d_filterKernel[7];
					tmpFilterValue += d_Signal[((x + 1)* signalWidth) + y] * d_filterKernel[8];
				}
			}

			// for each location apply the filter kernel
			for (uint32_t i = imageStarti; i < kernelIMax; i++) // assumes kernel af uneven squared size
			{
				for (uint32_t j = imageStartj; j < kernelJMax; j++)
				{
					float signalValue = d_Signal[((((imageStartx - 1) + (i - extraSubtractI))*signalWidth) + (imageStarty - 1)) + (j - extraSubtractY)];
					float kernelValue = d_filterKernel[j + (i*kernelDim)];

					tmpFilterValue += d_Signal[((((imageStartx - 1) + (i - extraSubtractI))*signalWidth) + (imageStarty - 1)) + (j - extraSubtractY)] * d_filterKernel[j + (i*kernelDim)];
				}
			}
		}

		d_result[(x*signalWidth) + y] = tmpFilterValue;
	}

}

__global__ void naive_custom_normalized_cross_correlation3D(
	float*				d_response,
	const float* 		d_original,
	const float* 		d_template,
	uint16_t			templateLength,
	uint16_t			templateChannels,
	uint32_t			signalLength,
	uint16_t			signalChannels,
	uint16_t*			d_signalLowerIndex
)
{
	// These values are stored in thread register
	// Make sure not to make more than 64 bytes of variables, as this is the max on the GPU GTX 1060!
	// Other GPU have other limits! - otherwise the data will be stored in a slow to read/write location
	// Which makes the computation time increase dramatically!
	//uint16_t numberOfIterations = (signalLength - templateLength) / (blockDim.x*gridDim.x);			// Number of iterations each thread has to go through
	uint16_t signalIndex = d_signalLowerIndex[blockIdx.y];
	float xcorr = 0;																		// Cross correlation between template and pixel area
	float varSignal = 0;																	// Variance Signal area
	float varTemp = 0;																		// Variance template
	float avgSignal = 0;																	// Average Signal area
	float avgTemp = 0;																		// Average template
	uint32_t signalIndexOffset = threadIdx.x + (blockDim.x*blockIdx.x); //+ (counter*blockDim.x*gridDim.x);
																							// blockDim.y represents which of the template that the thread is working on, e.g. blockDim.y = 0 equals the first template, 1 equals the seconds ...
	if (signalIndexOffset < (signalLength - templateLength))
	{
																							/* TEMPLATE RELATED */
																							// Inlined mean calculation of template
		for (uint16_t i = 0; i<templateLength; i++)
			for (uint16_t j = 0; j<templateChannels; j++) {
				avgTemp += d_template[(i * templateChannels) + j + (blockIdx.y*templateLength*templateChannels)]; // Computes average
																												  //avgTemp += d_template[(i * templateChannels) + j]; // Computes average
			}
		avgTemp = avgTemp / (templateChannels*templateLength);

		// Compute variance of template
		for (uint16_t i = 0; i < templateLength; i++) // Cross correlation with template
			for (uint16_t j = 0; j < templateChannels; j++) {

				float tr = d_template[i * templateChannels + j + (blockIdx.y*templateLength*templateChannels)] - avgTemp;
				//float tr = d_template[i * templateChannels + j] - avgTemp;
				varTemp += (tr*tr);
			}

	
		// Computes mean of image area
			// avgSignal = mean(signal, j, 0, w, wt, ht);

			// Inlined mean calculation
		avgSignal = 0;
		for (uint32_t i = signalIndexOffset; i < templateLength + signalIndexOffset; i++)
			for (uint32_t j = signalIndex; j < templateChannels + signalIndex; j++) {
				avgSignal += d_original[(i * signalChannels) + j]; // Computes average
			}
		avgSignal = avgSignal / (templateChannels*templateLength);

		// Clear variance and cross correlation
		xcorr = 0;
		varSignal = 0;

		// Computes cross correlation and variance
		for (uint32_t i = 0; i < templateLength; i++) // Cross correlation with template
			for (uint32_t j = 0; j < templateChannels; j++) {
				//float signalValue = d_original[(((x + signalIndexOffset)*templateChannels) + y + d_signalLowerIndex[blockDim.y])];
				//float temp = d_template[(x*templateChannels) + y + (blockDim.y*templateLength*templateChannels)];

				float pr = d_original[(((i + signalIndexOffset)*signalChannels) + j + signalIndex)] - avgSignal;
				//float tr = temp - avgTemp;
				xcorr += ((pr) * (d_template[(i*templateChannels) + j + (blockIdx.y*templateLength*templateChannels)] - avgTemp));
				//xcorr += ((pr) * (d_template[(i*templateChannels) + j] - avgTemp));
				varSignal += ((pr) * (pr));
			}

		// Computes normalized cross correlation
		//T normxcorr = xcorr / sqrt(varSignal * varTemp);
		if (varTemp != 0)
		{
			d_response[signalIndexOffset + (signalLength*blockIdx.y)] = xcorr / sqrtf(varSignal * varTemp);
		}
		else
		{
			d_response[signalIndexOffset + (signalLength*blockIdx.y)] = 0;
		}
		//d_response[signalIndexOffset] = xcorr / sqrtf(varSignal * varTemp);
	}
}

__global__ void naive_custom_normalized_cross_correlation3D_STD(
	float*				d_response,
	const float* 		d_original,
	const float* 		d_template,
	uint16_t			templateLength,
	uint16_t			templateChannels,
	uint32_t			signalLength,
	uint16_t			signalChannels,
	uint16_t*			d_signalLowerIndex
)
{
	// These values are stored in thread register
	// Make sure not to make more than 64 bytes of variables, as this is the max on the GPU GTX 1060!
	// Other GPU have other limits! - otherwise the data will be stored in a slow to read/write location
	// Which makes the computation time increase dramatically!
	//unsigned short numberOfIterations = (signalLength - templateLength) / (blockDim.x*gridDim.x);			// Number of iterations each thread has to go through
	uint16_t signalIndex = d_signalLowerIndex[blockIdx.y];
	float xcorr = 0;																		// Cross correlation between template and pixel area
	float varSignal = 0;																	// Variance Signal area
	float varTemp = 0;																		// Variance template
	float avgSignal = 0;																	// Average Signal area
	float avgTemp = 0;																		// Average template
	uint32_t signalIndexOffset = threadIdx.x + (blockDim.x*blockIdx.x);
	//const signed short signalLowerIndexOld = signalLowerIndex;
	// blockDim.y represents which of the template that the thread is working on, e.g. blockDim.y = 0 equals the first template, 1 equals the seconds ..

	if (signalIndexOffset < (signalLength - templateLength))
	{
		for (unsigned short d = 0; d < ((NUMBER_OF_DRIFT_CHANNELS_HANDLED * 2) + 1); d++)
		{
			int16_t dataOffset = d - NUMBER_OF_DRIFT_CHANNELS_HANDLED;
			int16_t templateStartChannel = 0;
			int16_t templateEndChannel = templateChannels;
			int16_t dataEndChannel = templateChannels;
			int16_t signalLowerIndex = signed short(signalIndex);

			if ((signalIndex + templateChannels + dataOffset) <= DATA_CHANNELS && /* the data and template must be cropped ! */
				(int16_t(signalIndex) + dataOffset) >= 0)
			{
				signalLowerIndex = signalIndex + dataOffset;
			}
			else
			{
				if ((int16_t(signalIndex) + dataOffset) < 0)
				{
					templateStartChannel -= dataOffset; // Increment
					dataEndChannel -= 1;
					signalLowerIndex = 0;
					//templateEndChannel += dataOffset; // This will decrement!!
				}
				else if ((int16_t(signalIndex) + templateChannels + dataOffset) > DATA_CHANNELS)
				{
					//templateStartChannel -= dataOffset; // this will increment, as d will always be negative here!!
					signalLowerIndex = signalIndex + dataOffset;
					dataEndChannel -= 1;
					templateEndChannel -= dataOffset; // This will decrement!!
				}
			}


			/* TEMPLATE RELATED */
			// Inlined mean calculation of template
			avgTemp = 0;

			for (uint16_t i = 0; i < templateLength; i++)
				for (uint16_t j = templateStartChannel; j < templateStartChannel + (templateEndChannel - templateStartChannel); j++) {
					avgTemp += d_template[(i * templateChannels) + j + (blockIdx.y*templateLength*templateChannels)]; // Computes average
				}
			avgTemp = avgTemp / ((templateEndChannel - templateStartChannel)*templateLength);

			// Compute variance of template
			varTemp = 0;
			for (uint16_t i = 0; i < templateLength; i++) // Cross correlation with template
				for (uint16_t j = templateStartChannel; j < templateEndChannel; j++) {
					float tr = d_template[i * templateChannels + j + (blockIdx.y*templateLength*templateChannels)] - avgTemp;
					//float tr = d_template[i * templateChannels + j] - avgTemp;
					varTemp += (tr*tr);
				}

			/* SIGNAL AND TEMPLATE RELATED */
			// Computes mean of image area
			// avgSignal = mean(signal, j, 0, w, wt, ht);

			// Inlined mean calculation
			avgSignal = 0;
			for (uint32_t i = signalIndexOffset; i < templateLength + signalIndexOffset; i++)
				for (uint32_t j = signalLowerIndex; j < (templateEndChannel - templateStartChannel) + signalLowerIndex; j++) {
					avgSignal += d_original[(i * signalChannels) + j]; // Computes average
				}
			avgSignal = avgSignal / ((templateEndChannel - templateStartChannel)*templateLength);

			// Clear variance and cross correlation

			xcorr = 0;
			varSignal = 0;

			// Computes cross correlation and variance
			for (uint32_t i = 0; i < templateLength; i++) // Cross correlation with template
				for (uint32_t j = 0; j < dataEndChannel; j++) {
					//float signalValue = d_original[(((x + signalIndexOffset)*templateChannels) + y + d_signalLowerIndex[blockDim.y])];
					//float temp = d_template[(x*templateChannels) + y + (blockDim.y*templateLength*templateChannels)];

					float pr = d_original[(((i + signalIndexOffset)*signalChannels) + j + signalLowerIndex)] - avgSignal;
					//float tr = temp - avgTemp;
					xcorr += ((pr) * (d_template[(i*templateChannels) + j + (blockIdx.y*templateLength*templateChannels) + templateStartChannel] - avgTemp));
					//xcorr += ((pr) * (d_template[(i*templateChannels) + j] - avgTemp));
					varSignal += ((pr) * (pr));
				}

			// Computes normalized cross correlation
			//T normxcorr = xcorr / sqrt(varSignal * varTemp);
			if (d > 0)
			{
				float currentValue = xcorr / sqrtf(varSignal * varTemp);
				if (currentValue > d_response[signalIndexOffset + (((signalLength - templateLength) + 1)*blockIdx.y)])
				{
					d_response[signalIndexOffset + (((signalLength - templateLength) + 1)*blockIdx.y)] = currentValue;
				}
			}
			else
			{
				d_response[signalIndexOffset + (((signalLength - templateLength) + 1)*blockIdx.y)] = xcorr / sqrtf(varSignal * varTemp);
				//d_response[signalIndexOffset] = xcorr / sqrtf(varSignal * varTemp);
			}

		}
	}
}


__global__ void naive_GPU_FindValuesAboveThreshold3DPredict(
	char*				d_response,
	const float* 		d_signal,
	const float* 		d_threshold,
	uint32_t            signalLength,
	uint16_t			templateLength
)
{
	uint32_t index = threadIdx.x + (blockDim.x*blockIdx.x);
	uint16_t templateId = blockIdx.y;

	if (index < (signalLength - templateLength))
	{
		if (d_signal[index + (templateId*signalLength)] >= d_threshold[templateId])
		{
			d_response[index + (templateId*signalLength)] = 1;
		}
		else
		{
			d_response[index + (templateId*signalLength)] = 0;
		}
	}

}

__global__ void naive_GPU_FindValuesAboveThreshold3D(
	char*				d_response,
	const float* 				d_signal,
	float 				threshold,
	uint32_t            signalLength,
	uint16_t			templateLength
)
{
	uint32_t index = threadIdx.x + (blockDim.x*blockIdx.x);
	uint16_t templateId = blockIdx.y;

	if (index < (signalLength-templateLength))
	{
		if (d_signal[index + (templateId*signalLength)] >= threshold)
		{
			d_response[index + (templateId*signalLength)] = 1;
		}
		else
		{
			d_response[index + (templateId*signalLength)] = 0;
		}
	}

}

__global__ void naive_GPU_FindPeaks3D(
	const float* 				d_signal,
	char* 				aboveThresholdindicator,
	uint32_t			signalLength,
	uint16_t			templateLength
)
{
	uint32_t index = threadIdx.x + (blockDim.x*blockIdx.x);
	uint16_t templateId = blockIdx.y;

	if (index < (signalLength - templateLength))
	{

		// Assign first and last element first
		if (index > 1 || index < ((signalLength - templateLength) - 1))
		{
			if (aboveThresholdindicator[index + (templateId*signalLength)] > 0)
			{

				if (d_signal[index + (templateId*signalLength)] > d_signal[index + (templateId*signalLength) - 1] && d_signal[index + (templateId*signalLength)] >= d_signal[index + (templateId*signalLength) + 1])
				{
					//numberOfPeaks++;
				}
				else
				{
					aboveThresholdindicator[index + (templateId*signalLength)] = 0;
				}
			}
		}
		else
		{
			if (index < 1)
			{
				if (d_signal[index + (templateId*signalLength)] > d_signal[index + (templateId*signalLength) + 1] && aboveThresholdindicator[index + (templateId*signalLength)] > 0)
				{
					//numberOfPeaks++;
				}
				else
				{
					aboveThresholdindicator[index + (templateId*signalLength)] = 0;
				}
			}

			if (index > ((signalLength - templateLength) - 2))
			{
				if (d_signal[index + (templateId*signalLength)] > d_signal[index + (templateId*signalLength) - 1] && aboveThresholdindicator[index + (templateId*signalLength)] > 0)
				{
					//numberOfPeaks++;
				}
				else
				{
					aboveThresholdindicator[index + (templateId*signalLength)] = 0;
				}
			}
		}
	}
}

__global__ void naive_GPU_MakesFoundTimes3D(
	uint32_t* 			dev_result,
	char* 				aboveThresholdindicator,
	uint32_t			signalLength,
	uint32_t			maxDimOfResult,
	uint32_t*			dev_counter,
	uint16_t			templateLength
)
{

	uint32_t index = threadIdx.x + (blockDim.x*blockIdx.x);
	uint16_t templateId = blockIdx.y;

	if (index < (signalLength - templateLength))
	{
		// Assign first and last element first
		if (aboveThresholdindicator[index + (templateId*signalLength)] > 0)
		{
			register uint32_t i = atomicAdd(&dev_counter[templateId], 1);
			if (i < maxDimOfResult)
			{
				dev_result[i + (templateId*maxDimOfResult)] = index;
			}
		}
	}
}

__global__ void naive_compare_with_truth_table3D(
	uint32_t*		  d_TPCounter,
	uint32_t*         d_truthTable,
	uint32_t* 		  d_estimationTable,
	uint32_t* 		  d_truthTableStartInd,
	uint32_t* 		  d_truthTableStartSize,
	uint32_t*         d_estimationTableSize,
	uint16_t*         d_peakOffset,
	uint32_t		  maxDimOfResult
)
{
	bool TP = false;
	uint32_t offsetSpike = 0;
	uint32_t I = threadIdx.x + (blockIdx.x*blockDim.x); // e.g threadIdx.x = 2, blockIdx.x = 4, blockDim.c = 1024 --> (4*1024)+2 = 4098
	uint16_t templateId = blockIdx.y;

	if (TEMPLATE_CROPPED_LENGTH > ((d_peakOffset[templateId] * 2) + 1))
	{
		offsetSpike = d_peakOffset[templateId];
	}
	else
	{
		offsetSpike = (d_peakOffset[templateId] / 2);
	}


	if (I < d_estimationTableSize[templateId])
	{
		bool timeStampLocated = false;

		for (uint32_t i = d_truthTableStartInd[templateId]; i < (d_truthTableStartInd[templateId] + d_truthTableStartSize[templateId]); i++)
		{		
			if ((d_estimationTable[I + (templateId*maxDimOfResult)] + offsetSpike) == (d_truthTable[i] - 1))
			{
				TP = true;
				timeStampLocated = true;
				break;
			}
		}

		if (!timeStampLocated && ACCEPTED_TIMELINE_SLACK > 0)
		{
			for (uint32_t Y = 1; Y <= ACCEPTED_TIMELINE_SLACK; Y++)
			{
				for (uint32_t i = d_truthTableStartInd[templateId]; i < (d_truthTableStartInd[templateId] + d_truthTableStartSize[templateId]); i++)
				{
					if ((d_estimationTable[I + (templateId*maxDimOfResult)] + offsetSpike) == ((d_truthTable[i] - 1) - Y))
					{
						TP = true;
						timeStampLocated = true;
						break;
					}
				}

				if (timeStampLocated)
				{
					break;
				}

				if (!timeStampLocated)
				{
					for (uint32_t i = d_truthTableStartInd[templateId]; i < (d_truthTableStartInd[templateId] + d_truthTableStartSize[templateId]); i++)
					{
						if ((d_estimationTable[I + (templateId*maxDimOfResult)] + offsetSpike) == ((d_truthTable[i] - 1) + Y))
						{
							TP = true;
							timeStampLocated = true;
							break;
						}
					}
				}

				if (timeStampLocated)
				{
					break;
				}
			}
		}
	}


	if (TP)
	{
		atomicAdd(&d_TPCounter[templateId], 1);
	}
}



extern "C" void PredictCUDA(const float *dev_signal, char *dev_aboveThreshold, uint32_t *dev_foundTimes, uint32_t *dev_foundTimesCounter,
	uint16_t templateLength, uint32_t signalLength, uint16_t numberOfTemplates, float *dev_threshold)
{
	uint32_t GridXSize = signalLength / MAXIMUM_NUMBER_OF_THREADS;

	if (signalLength % MAXIMUM_NUMBER_OF_THREADS != 0)
	{
		GridXSize++;
	}

	const dim3 blockSize(MAXIMUM_NUMBER_OF_THREADS, 1, 1);
	const dim3 gridsize(GridXSize, numberOfTemplates, 1);

	naive_GPU_FindValuesAboveThreshold3DPredict << <gridsize, blockSize >> > (dev_aboveThreshold, dev_signal, dev_threshold, signalLength, templateLength);
	naive_GPU_FindPeaks3D << <gridsize, blockSize >> > (dev_signal, dev_aboveThreshold, signalLength, templateLength);
	naive_GPU_MakesFoundTimes3D << <gridsize, blockSize >> > (dev_foundTimes, dev_aboveThreshold, signalLength, (uint32_t)MAXIMUM_PREDICTION_SAMPLES, dev_foundTimesCounter, templateLength);
}

extern "C" void TrainPart1CUDA(const float *dev_signal, char *dev_aboveThreshold, uint32_t *dev_foundTimes, uint32_t *dev_foundTimesCounter, 
							   uint32_t *dev_TPCounter, uint16_t *dev_peaksOffsets, uint32_t *devTruthTable, uint32_t *devTruthTableSize,
							   uint32_t *devTruthTableStartInd, uint16_t templateLength, uint32_t signalLength, uint16_t numberOfTemplates, float threshold)
{

	uint32_t GridXSize = signalLength / MAXIMUM_NUMBER_OF_THREADS;

	if (signalLength % MAXIMUM_NUMBER_OF_THREADS != 0)
	{
		GridXSize++;
	}

	const dim3 blockSize(MAXIMUM_NUMBER_OF_THREADS, 1, 1);
	const dim3 gridsize(GridXSize, numberOfTemplates, 1);

	naive_GPU_FindValuesAboveThreshold3D << <gridsize, blockSize >> > (dev_aboveThreshold, dev_signal, threshold, signalLength, templateLength);
	naive_GPU_FindPeaks3D << <gridsize, blockSize >> > (dev_signal, dev_aboveThreshold, signalLength, templateLength);
	naive_GPU_MakesFoundTimes3D << <gridsize, blockSize >> > (dev_foundTimes, dev_aboveThreshold, signalLength, (uint32_t)MAXIMUM_PREDICTION_SAMPLES, dev_foundTimesCounter, templateLength);
	
	const dim3 blockSizeCompare(MAXIMUM_NUMBER_OF_THREADS_COMPARING, 1, 1);

	GridXSize = MAXIMUM_PREDICTION_SAMPLES / MAXIMUM_NUMBER_OF_THREADS_COMPARING;
	if (MAXIMUM_PREDICTION_SAMPLES % MAXIMUM_NUMBER_OF_THREADS_COMPARING != 0)
	{
		GridXSize++;
	}
	const dim3 gridsizeCompare(GridXSize, numberOfTemplates, 1);

	naive_compare_with_truth_table3D << <gridsizeCompare, blockSizeCompare >> > (dev_TPCounter, devTruthTable, dev_foundTimes, devTruthTableStartInd, devTruthTableSize, dev_foundTimesCounter, dev_peaksOffsets, (uint32_t)MAXIMUM_PREDICTION_SAMPLES);
	

}

extern "C" void NXCOR_CUDA_3D(float *dev_result, const float *dev_templates, const float *dev_signal, uint16_t templateLength, uint16_t templateChannels, uint32_t signalLength, uint16_t signalChannels, uint16_t numberOfTemplates, uint16_t* dev_signalLowerIndex)
{
	hipError_t cudaStatus;
	uint32_t GridXSize = signalLength / MAXIMUM_NUMBER_OF_THREADS;

	if (signalLength % MAXIMUM_NUMBER_OF_THREADS != 0)
	{
		GridXSize++;
	}

	const dim3 blockSize(MAXIMUM_NUMBER_OF_THREADS, 1, 1);
	const dim3 gridsize(GridXSize, numberOfTemplates, 1);

	naive_custom_normalized_cross_correlation3D << <gridsize, blockSize >> >(dev_result, dev_signal, dev_templates, templateLength, templateChannels, signalLength, signalChannels, dev_signalLowerIndex);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "naive_custom_normalized_cross_correlation3D launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

}

extern "C" void NXCOR_CUDA_3D_Drift(float *dev_result, const float *dev_templates, const float *dev_signal, uint16_t templateLength, uint16_t templateChannels, uint32_t signalLength,
	uint16_t signalChannels, uint16_t numberOfTemplates, uint16_t* dev_signalLowerIndex)
{
	hipError_t cudaStatus;
	uint32_t GridXSize = signalLength / MAXIMUM_NUMBER_OF_THREADS_DRIFT_HANDLING;

	if (signalLength % MAXIMUM_NUMBER_OF_THREADS_DRIFT_HANDLING != 0)
	{
		GridXSize++;
	}

	const dim3 blockSize(MAXIMUM_NUMBER_OF_THREADS_DRIFT_HANDLING, 1, 1);
	const dim3 gridsize(GridXSize, numberOfTemplates, 1);

	naive_custom_normalized_cross_correlation3D_STD << <gridsize, blockSize >> >(dev_result, dev_signal, dev_templates, templateLength, templateChannels, signalLength, signalChannels, dev_signalLowerIndex);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "naive_custom_normalized_cross_correlation3D_STD launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

}


extern "C" void KernelFilterWithCudaV2(const float *dev_kernel, const float *dev_signal, float *dev_result, uint16_t templateChannels, uint16_t kernelDim, uint32_t signalLength)
{
	hipError_t cudaStatus;
	// Launch a kernel on the GPU with one thread for each element.
	int xBlocks = MAXIMUM_NUMBER_OF_THREADS / templateChannels;
	int xGrids = signalLength / xBlocks;
	const dim3 blockSize(xBlocks, templateChannels, 1);
	
	if (signalLength % xBlocks != 0)
	{
		xGrids++;
	}

	const dim3 gridsize(xGrids, 1, 1);

	runFilterReplicateGPU << <gridsize, blockSize >> >(dev_result, dev_signal, dev_kernel, kernelDim, signalLength, templateChannels);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "runFilterReplicateGPU launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
}

extern "C" void ChannelFilterWithCuda(float *dev_result, float *dev_signal, float *dev_resultInt, float* dev_coeffsA, float* dev_coeffsB, uint16_t signalWidth, uint32_t signalLength)
{
	hipError_t cudaStatus;
	const dim3 blockSize(signalWidth, 1, 1);
	//const dim3 blockSize(1, 1, 1);
	const dim3 gridsize(1, 1, 1);
	//runChannelFilterGPU << <gridsize, blockSize >> >(dev_result, dev_resultInt, dev_signal, dev_coeffsA, dev_coeffsB, signalWidth, signalLength);

	runChannelFilterForwardGPU << <gridsize, blockSize >> >(dev_resultInt, dev_signal, dev_coeffsA, dev_coeffsB, signalWidth, signalLength);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "runChannelFilterForwardGPU launch failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	runChannelFilterReverseGPU << <gridsize, blockSize >> >(dev_result, dev_resultInt, dev_coeffsA, dev_coeffsB, signalWidth, signalLength);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "runChannelFilterReverseGPU launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
}

extern "C" hipError_t SelectCUDA_GPU_Unit(void)
{
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	return cudaStatus;
}

extern "C" hipError_t AllocateCUDAData(float **dev_pointer, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)dev_pointer, (length*width) * bytesInValue);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t AllocateCUDADataChar(char **dev_pointer, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)dev_pointer, (length*width) * bytesInValue);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t AllocateCUDADataU16(uint16_t **dev_pointer, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)dev_pointer, (length*width) * bytesInValue);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t AllocateCUDADataU32(uint32_t **dev_pointer, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)dev_pointer, (length*width) * bytesInValue);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t MemCpyCUDAData(float *dev_pointer, float *host_pointer, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_pointer, host_pointer, ((length*width) * bytesInValue), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to device failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t MemCpyCUDADataU16(uint16_t *dev_pointer, uint16_t *host_pointer, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_pointer, host_pointer, ((length*width) * bytesInValue), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to device failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t MemCpyCUDADataU32(uint32_t *dev_pointer, uint32_t *host_pointer, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_pointer, host_pointer, ((length*width) * bytesInValue), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to device failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t RetreiveResults(float *dev_result, float *result, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result, dev_result, (width*length) * bytesInValue, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to host failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t RetreiveResultsU32(uint32_t *dev_result, uint32_t *result, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result, dev_result, (width*length) * bytesInValue, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to host failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t CheckForCudaError(void)
{
	hipError_t cudaStatus;
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
		return cudaStatus;
	}

	return cudaStatus;
}

extern "C" void CleanUpCudaForSpikeDet(float *dev_kernel)
{
	hipFree(dev_kernel);
}

extern "C" void CleanUpCudaForSpikeDetU16(uint16_t *dev_kernel)
{
	hipFree(dev_kernel);
}

extern "C" void CleanUpCudaForSpikeDetU32(uint32_t *dev_kernel)
{
	hipFree(dev_kernel);
}

extern "C" void CleanUpCudaForSpikeDetChar(char *dev_kernel)
{
	hipFree(dev_kernel);
}

#endif